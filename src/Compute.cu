#include "hip/hip_runtime.h"
#include "Compute.hpp"
#include "Image.hpp"
#include "logo.h"
#include <iostream>
#include <cmath> // For sqrtf and std::pow

struct Lab
{
    double L, a, b;
};

template <typename T>
__device__ void mySwapCuda(T& a, T& b) {
    T temp = a;
    a = b;
    b = temp;
}

template <typename T>
__device__ T myMinCuda(T a, T b) {
    return a < b ? a : b;
}

template <typename T>
__device__ T myMaxCuda(T a, T b) {
    return a > b ? a : b;
}

__device__ double sRGBToLinear_cuda(double c) {
    if (c <= 0.04045)
        return c / 12.92;
    else
        return pow((c + 0.055) / 1.055, 2.4);
}

// Function for XYZ to Lab conversion
__device__ double f_xyz_to_lab_cuda(double t) {
    const double epsilon = 0.008856; // (6/29)^3
    const double kappa = 903.3;      // (29/3)^3

    if (t > epsilon)
        return cbrt(t); // Cube root
    else
        return (kappa * t + 16.0) / 116.0;
}

// Function to convert RGB to XYZ
__device__ void rgbToXyz_cuda(const rgb8& rgb, double& X, double& Y, double& Z) {
    // Normalize RGB values between 0 and 1
    double r = sRGBToLinear_cuda(rgb.r / 255.0);
    double g = sRGBToLinear_cuda(rgb.g / 255.0);
    double b = sRGBToLinear_cuda(rgb.b / 255.0);

    // sRGB D65 conversion matrix
    X = r * 0.4124564 + g * 0.3575761 + b * 0.1804375;
    Y = r * 0.2126729 + g * 0.7151522 + b * 0.0721750;
    Z = r * 0.0193339 + g * 0.1191920 + b * 0.9503041;
}

// Function to convert XYZ to Lab
__device__ Lab xyzToLab_cuda(double X, double Y, double Z) {
    // Reference white D65
    const double Xr = 0.95047;
    const double Yr = 1.00000;
    const double Zr = 1.08883;

    // Normalize by reference white
    double x = X / Xr;
    double y = Y / Yr;
    double z = Z / Zr;

    // Apply f(t) function
    double fx = f_xyz_to_lab_cuda(x);
    double fy = f_xyz_to_lab_cuda(y);
    double fz = f_xyz_to_lab_cuda(z);

    Lab lab;
    lab.L = 116.0 * fy - 16.0;
    lab.a = 500.0 * (fx - fy);
    lab.b = 200.0 * (fy - fz);

    return lab;
}

// Function to convert RGB to Lab
__device__ Lab rgbToLab_cuda(const rgb8& rgb) {
    double X, Y, Z;
    rgbToXyz_cuda(rgb, X, Y, Z);
    return xyzToLab_cuda(X, Y, Z);
}

// Function to compute ΔE (CIE76) between two Lab colors
__device__ double deltaE_cuda(const Lab& lab1, const Lab& lab2) {
    double dL = lab1.L - lab2.L;
    double da = lab1.a - lab2.a;
    double db = lab1.b - lab2.b;
    return sqrt(dL * dL + da * da + db * db);
}

__device__ double background_estimation(ImageView<rgb8> in, ImageView<rgb8> device_background, ImageView<rgb8> device_candidate, ImageView<uint8_t> pixel_time_counter)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);
    rgb8* bg_pixel = (rgb8*)((std::byte*)device_background.buffer + y * device_background.stride);
    rgb8* candidate_pixel = (rgb8*)((std::byte*)device_candidate.buffer + y * device_candidate.stride);

    // Moyenne locale sur les pixels voisins pour une estimation plus stable
    int sumR = 0, sumG = 0, sumB = 0, count = 0;
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && nx < in.width && ny >= 0 && ny < in.height) {
                rgb8 neighbor_pixel = *((rgb8*)((std::byte*)in.buffer + ny * in.stride) + nx);
                sumR += neighbor_pixel.r;
                sumG += neighbor_pixel.g;
                sumB += neighbor_pixel.b;
                count++;
            }
        }
    }
    rgb8 mean_pixel = {static_cast<uint8_t>(sumR / count),
                       static_cast<uint8_t>(sumG / count),
                       static_cast<uint8_t>(sumB / count)};

    // Calcul de la distance ΔE entre le pixel de fond et la moyenne locale
    double distance = deltaE_cuda(rgbToLab_cuda(pixel[x]), rgbToLab_cuda(bg_pixel[x]));
    bool match = distance < 2;

    uint8_t *time = (uint8_t*)((std::byte*)pixel_time_counter.buffer + y * pixel_time_counter.stride);

    if (!match) {
        if (time[x] == 0) {
            candidate_pixel[x] = pixel[x];//mean_pixel;
            time[x] += 1;
        } else if (time[x] < 100) {
            candidate_pixel[x].r = (candidate_pixel[x].r + mean_pixel.r) / 2;
            candidate_pixel[x].g = (candidate_pixel[x].g + mean_pixel.g) / 2;
            candidate_pixel[x].b = (candidate_pixel[x].b + mean_pixel.b) / 2;
            time[x] += 1;
        } else {
            mySwapCuda(bg_pixel[x].r, mean_pixel.r);
            mySwapCuda(bg_pixel[x].g, mean_pixel.g);
            mySwapCuda(bg_pixel[x].b, mean_pixel.b);
            time[x] = 0;
        }
    } else {
        // Mise à jour progressive du fond avec interpolation pour un lissage
        bg_pixel[x].r = static_cast<uint8_t>(bg_pixel[x].r * 0.9 + mean_pixel.r * 0.1);
        bg_pixel[x].g = static_cast<uint8_t>(bg_pixel[x].g * 0.9 + mean_pixel.g * 0.1);
        bg_pixel[x].b = static_cast<uint8_t>(bg_pixel[x].b * 0.9 + mean_pixel.b * 0.1);
        time[x] = 0;
    }

    return distance;
}

__device__ void apply_filter(ImageView<rgb8> in, double distance) {

    const double distanceMultiplier = 2.8;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in.width || y >= in.height)
        return;

    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);

    pixel[x].r = static_cast<uint8_t>(myMinCuda(255.0, distance * distanceMultiplier));
}

__device__ void morphological(
    ImageView<rgb8> in,
    ImageView<rgb8> copy,
    const int* kernel,
    int radius,
    int diameter,
    bool erode)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Verify if (x, y) is within bounds
    if (x >= radius && x < (in.width - radius) && y >= radius && y < (in.height - radius)) {
        rgb8* pixel = (rgb8*)((std::byte*)copy.buffer + y * copy.stride);

        uint8_t new_value = (erode) ? 255 : 0;
        for (int ky = 0; ky < diameter; ++ky) {
            for (int kx = 0; kx < diameter; ++kx) {
                if (kernel[ky * diameter + kx] == 1) {
                    int ny = y + ky - radius;
                    int nx = x + kx - radius;

                    rgb8* kernel_pixel = (rgb8*)((std::byte*)in.buffer + ny * in.stride);
                    if (erode) {
                        new_value = myMinCuda(new_value, kernel_pixel[nx].r);
                    }
                    else {
                        new_value = myMaxCuda(new_value, kernel_pixel[nx].r);
                    }
                }
            }
        }
        pixel[x].r = new_value;
    }
}

__global__ void morphologicalOpening(
    ImageView<rgb8> in,
    ImageView<rgb8> copy,
    const int* diskKernel,
    int radius,
    int diameter,
    bool erode)
{
    // Step 1: Erosion
    morphological(in, copy, diskKernel, radius, diameter, erode);
    // Step 2: Dilation
    //morphological(copy, in, diskKernel, radius, diameter, false);
}

__device__ void hysteresis_threshold_process(ImageView<rgb8> in, int lowThreshold, int highThreshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in.width || y >= in.height)
        return;

    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);
    int intensity = pixel[x].r;

    if (intensity >= highThreshold) {
        //in.buffer[y * in.width + x] = {255, 255, 255};
        pixel[x].r = 255;
        pixel[x].g = 255;
        pixel[x].b = 255;
    } else if (intensity < lowThreshold) {
        //in.buffer[y * in.width + x] = {0, 0, 0};
        pixel[x].r = 0;
        pixel[x].g = 0;
        pixel[x].b = 0;
    } else {
        //in.buffer[y * in.width + x] = {127, 127, 127};
        pixel[x].r = 127;
        pixel[x].g = 127;
        pixel[x].b = 127;
    }
}

__device__ void propagate_edges(ImageView<rgb8> in, int lowThreshold, int highThreshold, bool* hasChanged) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in.width || y >= in.height)
        return;

    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);

    if (pixel[x].r == 255) {
        // Vérifier si un voisin est un bord fort
        for (int dy = -1; dy <= 1 && !*hasChanged; dy++) {
            for (int dx = -1; dx <= 1; dx++) {
                if (dx == 0 && dy == 0) continue;

                int neighborX = x + dx;
                int neighborY = y + dy;

                if (neighborX >= 0 && neighborX < in.width && neighborY >= 0 && neighborY < in.height) {
                    rgb8* neighborPixel = (rgb8*)((std::byte*)in.buffer + neighborY * in.stride);
                    int neighborIntensity = neighborPixel[neighborX].r;
                    if (neighborIntensity >= lowThreshold && neighborIntensity < highThreshold && neighborPixel[neighborX].r != 255) { // Bord fort
                        neighborPixel[neighborX] = {255, 255, 255};
                        *hasChanged = true;
                    }
                }
            }
        }
    }
}

__global__ void hysteresis(ImageView<rgb8> in, int lowThreshold, int highThreshold) {
    hysteresis_threshold_process(in, lowThreshold, highThreshold);

    // Propagate edges
    bool updated;
    do {
        updated = false;
        propagate_edges(in, lowThreshold, highThreshold, &updated);
    } while (updated);

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in.width || y >= in.height)
        return;

    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);

    if (pixel[x].r == 127) {
        pixel[x].r = 0;
        pixel[x].g = 0;
        pixel[x].b = 0;
    }
}

__global__ void background_estimation_process(
    ImageView<rgb8> in,
    ImageView<rgb8> device_background,
    ImageView<rgb8> device_candidate,
    ImageView<uint8_t> pixel_time_counter,
    ImageView<rgb8> copy,
    const int* diskKernel,
    int radius,
    int diameter)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in.width || y >= in.height)
        return;

    double distance = background_estimation(in, device_background, device_candidate, pixel_time_counter);
    
    const double distanceMultiplier = 2.8;
    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);
    pixel[x].r = static_cast<uint8_t>(myMinCuda(255.0, distance * distanceMultiplier));
    
    rgb8* pixel_copy = (rgb8*)((std::byte*)copy.buffer + y * copy.stride);
    pixel_copy[x].r = static_cast<uint8_t>(myMinCuda(255.0, distance * distanceMultiplier));
    
    //apply_filter(in, distance);

    //morphologicalOpening(in, copy, diskKernel, radius, diameter);
}

__global__ void applyRedMask_cuda(ImageView<rgb8> in, ImageView<rgb8> initialPixels){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in.width || y >= in.height)
        return;

    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);
    rgb8* initial_pixel = (rgb8*)((std::byte*)initialPixels.buffer + y * initialPixels.stride);

    if (pixel[x].r > 0) {
        pixel[x].r = myMinCuda(255, static_cast<int>(initial_pixel[x].r + 0.5 * 255));
        pixel[x].g = initial_pixel[x].g;
        pixel[x].b = initial_pixel[x].b;
      }
    else {
        pixel[x].r = initial_pixel[x].r;
        pixel[x].g = initial_pixel[x].g;
        pixel[x].b = initial_pixel[x].b;
      }
}

void compute_cu(ImageView<rgb8> in)
{
    static Image<uint8_t> device_logo;
    static Image<uint8_t> pixel_time_counter;
    static Image<rgb8> device_background;
    static Image<rgb8> device_candidate;

    dim3 block(16, 16);
    dim3 grid((in.width + block.x - 1) / block.x, (in.height + block.y - 1) / block.y);

    // Copy the logo to the device if it is not already there
    if (device_logo.buffer == nullptr)
    {
        device_logo = Image<uint8_t>(logo_width, logo_height, true);
        hipMemcpy2D(device_logo.buffer, device_logo.stride, logo_data, logo_width, logo_width, logo_height, hipMemcpyHostToDevice);
    }

    if (device_background.buffer == nullptr)
    {
        device_background = Image<rgb8>(in.width, in.height, true);
        hipMemcpy2D(device_background.buffer, device_background.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice);

        device_candidate = Image<rgb8>(in.width, in.height, true);
        hipMemcpy2D(device_candidate.buffer, device_candidate.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice);

        pixel_time_counter = Image<uint8_t>(in.width, in.height, true);
        hipMemset2D(pixel_time_counter.buffer, pixel_time_counter.stride, 0, in.width, in.height);
    }

    // Copy the input image to the device
    Image<rgb8> device_in(in.width, in.height, true);
    hipMemcpy2D(device_in.buffer, device_in.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice);

    //copy Initial Pixel
    Image<rgb8> Initialcopy(in.width, in.height, true);
    hipMemcpy2D(Initialcopy.buffer, Initialcopy.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice);


    // Create a copy of the input image for morphological operations
    Image<rgb8> copy(in.width, in.height, true);
    hipMemcpy2D(copy.buffer, copy.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyDeviceToDevice);

    // Compute the radius for the kernel
    int min_dimension = std::min(in.width, in.height);
    int ratio_disk = 1; // 1% of the smallest dimension
    int minradius = 5;
    int radius = std::max(minradius, (min_dimension / 100) * ratio_disk);

    // Create the disk kernel on the host
    int diameter = 2 * radius + 1;
    int kernel_size = diameter * diameter;
    int* h_diskKernel = new int[kernel_size];

    int center = radius;
    for (int i = 0; i < diameter; ++i) {
        for (int j = 0; j < diameter; ++j) {
            if (sqrtf((i - center) * (i - center) + (j - center) * (j - center)) <= radius) {
                h_diskKernel[i * diameter + j] = 1;
            }
            else {
                h_diskKernel[i * diameter + j] = 0;
            }
        }
    }

    // Allocate device memory for the kernel
    int* d_diskKernel;
    hipMalloc(&d_diskKernel, kernel_size * sizeof(int));
    hipMemcpy(d_diskKernel, h_diskKernel, kernel_size * sizeof(int), hipMemcpyHostToDevice);

    // Clean up host kernel memory
    delete[] h_diskKernel;

    // Launch the kernel
    background_estimation_process<<<grid, block>>>(
        device_in,
        device_background,
        device_candidate,
        pixel_time_counter,
        copy,
        d_diskKernel,
        radius,
        diameter);

    // Synchronize and check for errors
    hipDeviceSynchronize();

    morphologicalOpening<<<grid, block>>>(device_in, copy, d_diskKernel, radius, diameter, true);
    hipDeviceSynchronize();

    morphologicalOpening<<<grid, block>>>(device_in, copy, d_diskKernel, radius, diameter, false);
    hipDeviceSynchronize();
    //hipMemcpy2D(device_in.buffer, device_in.stride, copy.buffer, copy.stride, in.width * sizeof(rgb8), in.height, hipMemcpyDeviceToDevice);


    hysteresis<<<grid, block>>>(device_in, 10, 100);
    //propagate_edges_process<<<grid, block>>>(device_in, 20, 50);
    hipDeviceSynchronize();

    applyRedMask_cuda<<<grid, block>>>(device_in, Initialcopy);
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy2D(in.buffer, in.stride, device_in.buffer, device_in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyDeviceToHost);

    // Free device memory for the kernel
    hipFree(d_diskKernel);

    // Free other device memory if necessary
    // hipFree(device_in.buffer);
    // hipFree(copy.buffer);
}
