#include "hip/hip_runtime.h"
#include "Compute.hpp"
#include "Image.hpp"
#include "logo.h"
#include <iostream>

struct Lab
{
    double L, a, b;
};

template <typename T>
__device__ void mySwapCuda(T& a, T& b) {
    T temp = a;
    a = b;
    b = temp;
}

template <typename T>
__device__ T myMinCuda(T a, T b) {
    return a < b ? a : b;
}

template <typename T>
__device__ T myMaxCuda(T a, T b) {
    return a > b ? a : b;
}

__device__ double sRGBToLinear_cuda(double c) {
    if (c <= 0.04045)
        return c / 12.92;
    else
        return std::pow((c + 0.055) / 1.055, 2.4);
}

// Fonction auxiliaire pour la conversion XYZ -> Lab
__device__ double f_xyz_to_lab_cuda(double t) {
    const double epsilon = 0.008856; // (6/29)^3
    const double kappa = 903.3;      // (29/3)^3

    if (t > epsilon)
        return std::cbrt(t); // Racine cubique
    else
        return (kappa * t + 16.0) / 116.0;
}

// Fonction pour convertir RGB en XYZ
__device__ void rgbToXyz_cuda(const rgb8& rgb, double& X, double& Y, double& Z) {
    // Normalisation des valeurs RGB entre 0 et 1
    double r = sRGBToLinear_cuda(rgb.r / 255.0);
    double g = sRGBToLinear_cuda(rgb.g / 255.0);
    double b = sRGBToLinear_cuda(rgb.b / 255.0);

    // Matrice de conversion sRGB D65
    X = r * 0.4124564 + g * 0.3575761 + b * 0.1804375;
    Y = r * 0.2126729 + g * 0.7151522 + b * 0.0721750;
    Z = r * 0.0193339 + g * 0.1191920 + b * 0.9503041;
}
// Fonction pour convertir XYZ en Lab
__device__ Lab xyzToLab_cuda(double X, double Y, double Z) {
    // Blanc de référence D65
    const double Xr = 0.95047;
    const double Yr = 1.00000;
    const double Zr = 1.08883;

    // Normalisation par rapport au blanc de référence
    double x = X / Xr;
    double y = Y / Yr;
    double z = Z / Zr;

    // Application de la fonction f(t)
    double fx = f_xyz_to_lab_cuda(x);
    double fy = f_xyz_to_lab_cuda(y);
    double fz = f_xyz_to_lab_cuda(z);

    Lab lab;
    lab.L = 116.0 * fy - 16.0;
    lab.a = 500.0 * (fx - fy);
    lab.b = 200.0 * (fy - fz);

    return lab;
}

// Fonction pour convertir RGB en Lab
__device__ Lab rgbToLab_cuda(const rgb8& rgb) {
    double X, Y, Z;
    rgbToXyz_cuda(rgb, X, Y, Z);
    return xyzToLab_cuda(X, Y, Z);
}

// Fonction pour calculer la distance ΔE (CIE76) entre deux couleurs Lab
__device__ double deltaE_cuda(const Lab& lab1, const Lab& lab2) {
    double dL = lab1.L - lab2.L;
    double da = lab1.a - lab2.a;
    double db = lab1.b - lab2.b;
    return std::sqrt(dL * dL + da * da + db * db);
}

__device__ double background_estimation(ImageView<rgb8> in, ImageView<rgb8> device_background, ImageView<rgb8> device_candidate, ImageView<uint8_t> pixel_time_counter)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);
    rgb8* bg_pixel = (rgb8*)((std::byte*)device_background .buffer + y * device_background.stride);
    rgb8* candidate_pixel = (rgb8*)((std::byte*)device_candidate.buffer + y * device_candidate.stride);

    double distance = deltaE_cuda(rgbToLab_cuda(pixel[x]), rgbToLab_cuda(bg_pixel[x]));
    bool match = distance < 25;

    // std::cout << "aled: " << distance << std::endl;

    uint8_t *time = (uint8_t*)((std::byte*)pixel_time_counter.buffer + y * pixel_time_counter.stride);

    if (!match)
    {
        if(time[x] == 0)
        {
            candidate_pixel[x].r = pixel[x].r;
            candidate_pixel[x].g = pixel[x].g;
            candidate_pixel[x].b = pixel[x].b;
            time[x] += 1;
        }
        else if (time[x] < 25)
        {
            candidate_pixel[x].r = (candidate_pixel[x].r + pixel[x].r) / 2;
            candidate_pixel[x].g = (candidate_pixel[x].g + pixel[x].g) / 2;
            candidate_pixel[x].b = (candidate_pixel[x].b + pixel[x].b) / 2;
            time[x] += 1;
        }
        else
        {
            mySwapCuda(bg_pixel[x].r, candidate_pixel[x].r);
            mySwapCuda(bg_pixel[x].g, candidate_pixel[x].g);
            mySwapCuda(bg_pixel[x].b, candidate_pixel[x].b);
            time[x] = 0;        
        }
    }
    else {
        bg_pixel[x].r = (bg_pixel[x].r + pixel[x].r) / 2;
        bg_pixel[x].g = (bg_pixel[x].g + pixel[x].g) / 2;
        bg_pixel[x].b = (bg_pixel[x].b + pixel[x].b) / 2; 
        time[x] = 0;
    }
    return distance;
}

__device__ void apply_filter(ImageView<rgb8> in, double distance){

    const double distanceMultiplier = 2.8;
    const double threshold = 3.5;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in.width || y >= in.height)
        return;

    rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);

    // if (distance > threshold)
    // {
    //     uint8_t intensity = static_cast<uint8_t>(myMinCuda(distance * distanceMultiplier, 255.0));
    //     pixel[x].r = intensity;
    //     pixel[x].g = intensity;
    //     pixel[x].b = 0;
    // }
    // else
    // {
    //     pixel[x].r = 0;
    //     pixel[x].g = 0;
    //     pixel[x].b = 0;
    // }
             
    pixel[x].r = static_cast<uint8_t>(myMinCuda(255.0, distance * distanceMultiplier));

}


__device__ int* createDiskKernel_cuda(int radius) {
    int diameter = 2 * radius + 1;
    int* kernel = new int[diameter * diameter];
    int center = radius;
    
    for (int i = 0; i < diameter; ++i) {
        for (int j = 0; j < diameter; ++j) {
            if (sqrtf((i - center) * (i - center) + (j - center) * (j - center)) <= radius) {
                kernel[i * diameter + j] = 1;
            }
            else {
                kernel[i * diameter + j] = 0;
            }
        }
    }
    return kernel;
}

// __global__ void createKernelDevice(int radius, int* kernel) {
//     int diameter = 2 * radius + 1;
//     //int* kernel = new int[diameter * diameter];
//     int center = radius;
    
//     for (int i = 0; i < diameter; ++i) {
//         for (int j = 0; j < diameter; ++j) {
//             if (sqrtf((i - center) * (i - center) + (j - center) * (j - center)) <= radius) {
//                 kernel[i * diameter + j] = 1;
//             }
//             else {
//                 kernel[i * diameter + j] = 0;
//             }
//         }
//     }
// }

// __device__ Image<rgb8> clone(ImageView<rgb8> in)
// {
//     Image<rgb8> img = Image<rgb8>();
//     img.buffer = in.buffer;
//     img.width = in.width;
//     img.height = in.height;
//     img.stride = in.width;
//     //return img.clone();

// }

// Appliquer une opération d'érosion
__device__ void morphological(ImageView<rgb8> in, ImageView<rgb8> copy, const int* kernel, int radius, bool erode) {
    //Image<rgb8> copy = clone(in);  // Faire une copie temporaire de l'image pour éviter la corruption
    
    int diameter = 2 * radius + 1;
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Vérifiez si (x, y) est dans les limites et respecte le rayon
    if (x >= radius && x < (in.width - radius) && y >= radius && y < (in.height - radius)) {
        rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);
        
        uint8_t new_value = (erode) ? 255 : 0;
        for (int ky = 0; ky < diameter; ++ky) {
            for (int kx = 0; kx < diameter; ++kx) {

                if (kernel[ky * diameter + kx] == 1) {
                    int ny = y + ky - radius;
                    int nx = x + kx - radius;

                    rgb8* kernel_pixel = (rgb8*)((std::byte*)in.buffer + ny * in.stride);
                    if (erode){
                        new_value = myMinCuda(new_value, kernel_pixel[nx].r);
                    }
                    else {
                        new_value = myMaxCuda(new_value, kernel_pixel[nx].r);   
                    }                 
                }
            }
        }
        pixel = (rgb8*)((std::byte*)copy.buffer + y * copy.stride);
        pixel[x].r = new_value;
    }
}

__device__ void morphologicalOpening(ImageView<rgb8> in, ImageView<rgb8> copy, int minradius) {
    int min_dimension = myMinCuda(in.width, in.height);
    int ratio_disk = 1; // 1 % de la resolution de l'image
    int radius = myMaxCuda(minradius, (min_dimension / 100) * ratio_disk);
    // std::cout << "radius: " << radius << std::endl;
    // Créer un noyau en forme de disque avec le rayon calculé
    auto diskKernel = createDiskKernel_cuda(radius);
    // Étape 1 : Erosion
    morphological(in, copy, diskKernel, radius, true);
    // Étape 2 : Dilatation
    morphological(in, copy, diskKernel, radius, false);
}

__global__ void background_estimation_process(ImageView<rgb8> in, ImageView<rgb8> device_background, ImageView<rgb8> device_candidate, ImageView<uint8_t> pixel_time_counter, ImageView<rgb8> copy)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // dim3 block(16, 16);
    // dim3 grid((in.width + block.x - 1) / block.x, (in.height + block.y - 1) / block.y);

    if (x >= in.width || y >= in.height)
        return;

    double distance = background_estimation(in, device_background, device_candidate, pixel_time_counter);
    // if (distance > 1) {
    //     printf("distance: %f\n", distance);
    // }
    //rgb8* pixel = (rgb8*)((std::byte*)in.buffer + y * in.stride);

    //pixel[x].r = static_cast<uint8_t>(myMinCuda(distance * distanceMultiplier, 255.0));

    apply_filter(in, distance);


    // Étape 1 : Erosion
    

    //morphologicalOpening(in, copy, 3);

    //apply_filter<<<grid, block>>>(in, distance);
}


void compute_cu(ImageView<rgb8> in)
{
    static Image<uint8_t> device_logo;
    static Image<uint8_t> pixel_time_counter;
    static Image<rgb8> device_background;
    static Image<rgb8> device_candidate;

    dim3 block(16, 16);
    dim3 grid((in.width + block.x - 1) / block.x, (in.height + block.y - 1) / block.y);

    // Create a copy of the input image
    //----------
    
    // Copy the logo to the device if it is not already there
    if (device_logo.buffer == nullptr)
    {
        device_logo = Image<uint8_t>(logo_width, logo_height, true);
        hipMemcpy2D(device_logo.buffer, device_logo.stride, logo_data, logo_width, logo_width, logo_height, hipMemcpyHostToDevice);
    }

    if(device_background.buffer == nullptr)
    {
        device_background = Image<rgb8>(in.width, in.height, true);
        hipMemcpy2D(device_background.buffer, device_background.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice);

        device_candidate = Image<rgb8>(in.width, in.height, true);
        hipMemcpy2D(device_candidate.buffer, device_candidate.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice);

        pixel_time_counter = Image<uint8_t>(in.width, in.height, true);
        hipMemset2D(pixel_time_counter.buffer, pixel_time_counter.stride, 0, in.width, in.height);
    }

    // Copy the input image to the device
    Image<rgb8> device_in(in.width, in.height, true);
    hipMemcpy2D(device_in.buffer, device_in.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice);
    
    // mykernel<<<grid, block>>>(device_background, device_logo);

    Image<rgb8> copy(in.width, in.height, true); 
    hipMemcpy2D(copy.buffer, copy.stride, in.buffer, in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyDeviceToDevice);

    background_estimation_process<<<grid, block>>>(device_in, device_background, device_candidate, pixel_time_counter, copy);

    hipDeviceSynchronize();


    

    //hipMemcpy(device_in.buffer, device_in.stride, copy.buffer, in.width * in.height * sizeof(rgb8), hipMemcpyDeviceToDevice);

    // Copy the result back to the host
    //hipMemcpy2D(in.buffer, in.stride, device_background.buffer, device_background.stride, in.width * sizeof(rgb8), in.height, hipMemcpyDeviceToHost);
    hipMemcpy2D(in.buffer, in.stride, device_in.buffer, device_in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyDeviceToHost);
}