#include "hip/hip_runtime.h"
#include "Compute.hpp"
#include "Image.hpp"
#include <vector>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                          \
    do {                                                                          \
        hipError_t error = call;                                                 \
        if (error != hipSuccess) {                                               \
            std::cerr << "CUDA Error: " << hipGetErrorString(error)              \
                      << " at " << __FILE__ << ":" << __LINE__ << std::endl;      \
            exit(EXIT_FAILURE);                                                   \
        }                                                                         \
    } while (0)

struct Lab {
    float L;
    float a;
    float b;
};

template <typename T>
__device__ void mySwapCuda(T& a, T& b) {
    T temp = a;
    a = b;
    b = temp;
}
// Device functions for color space conversion and distance calculation
__device__ float sRGBToLinearGPU(float c) {
    return (c <= 0.04045f) ? (c / 12.92f) : powf((c + 0.055f) / 1.055f, 2.4f);
}

__device__ void rgbToXyzGPU(const rgb8& rgb, float& X, float& Y, float& Z) {
    float r = sRGBToLinearGPU(rgb.r / 255.0f);
    float g = sRGBToLinearGPU(rgb.g / 255.0f);
    float b = sRGBToLinearGPU(rgb.b / 255.0f);

    X = r * 0.4124564f + g * 0.3575761f + b * 0.1804375f;
    Y = r * 0.2126729f + g * 0.7151522f + b * 0.0721750f;
    Z = r * 0.0193339f + g * 0.1191920f + b * 0.9503041f;
}

__device__ Lab xyzToLabGPU(float X, float Y, float Z) {
    const float Xr = 0.95047f;
    const float Yr = 1.00000f;
    const float Zr = 1.08883f;

    float x = X / Xr;
    float y = Y / Yr;
    float z = Z / Zr;

    float fx = (x > 0.008856f) ? cbrtf(x) : ((903.3f * x + 16.0f) / 116.0f);
    float fy = (y > 0.008856f) ? cbrtf(y) : ((903.3f * y + 16.0f) / 116.0f);
    float fz = (z > 0.008856f) ? cbrtf(z) : ((903.3f * z + 16.0f) / 116.0f);

    Lab lab;
    lab.L = 116.0f * fy - 16.0f;
    lab.a = 500.0f * (fx - fy);
    lab.b = 200.0f * (fy - fz);

    return lab;
}

__device__ Lab rgbToLabGPU(const rgb8& rgb) {
    float X, Y, Z;
    rgbToXyzGPU(rgb, X, Y, Z);
    return xyzToLabGPU(X, Y, Z);
}

__device__ float deltaEGPU(const Lab& lab1, const Lab& lab2) {
    float dL = lab1.L - lab2.L;
    float da = lab1.a - lab2.a;
    float db = lab1.b - lab2.b;
    return sqrtf(dL * dL + da * da + db * db);
}

__device__ double mymin(const double a, const double b){
    if (a < b)
        return a;
    return b;
}


__global__ void back_ground_estimation(ImageView<rgb8> in, ImageView<rgb8> bg_value, ImageView<rgb8> candidate_value, int* time_matrix) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in.width || y >= in.height) return;

    int idx = y * in.width + x;


}

__global__ void applyFlow(ImageView<rgb8> in, ImageView<rgb8> bg_value, ImageView<rgb8> candidate_value, int* time_matrix)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    const double strictDistanceThreshold = 0.25;
    const double highlightDistanceMultiplier = 2.8; 

    if (x >= in.width || y >= in.height) return;

    rgb8 bg_pixel = bg_value.buffer[idx];
    rgb8 in_pixel = in.buffer[idx];
    rgb8 candidate_pixel = candidate_value.buffer[idx];

    Lab lab_in = rgbToLabGPU(in_pixel);
    Lab lab_bg = rgbToLabGPU(bg_pixel);

    double distance = deltaEGPU(lab_in, lab_bg);
    int time = time_matrix[idx];
    bool match = distance < 25.0;
    // int time = 0;
    if (match) {
        time = 0;
        bg_pixel.r = in_pixel.r;
        bg_pixel.g = in_pixel.g;
        bg_pixel.b = in_pixel.b;
        bg_value.buffer[idx] = bg_pixel;
    } else {
        if (time == 0) {
            candidate_pixel.r = in_pixel.r;
            candidate_pixel.g = in_pixel.g;
            candidate_pixel.b = in_pixel.b;
            candidate_value.buffer[idx] = candidate_pixel;
            time ++;
        } else if (time < 100) {
            candidate_pixel.r = (candidate_pixel.r + in_pixel.r) / 2;
            candidate_pixel.g = (candidate_pixel.g + in_pixel.g) / 2;
            candidate_pixel.b = (candidate_pixel.b + in_pixel.b) / 2;
            candidate_value.buffer[idx] = candidate_pixel;
            time++;
        } else {
            mySwapCuda(bg_pixel, candidate_pixel);
            bg_value.buffer[idx] = bg_pixel;
            candidate_value.buffer[idx] = candidate_pixel;
            time = 0;
        }
    }
    time_matrix[y * in.width + x] = time;

    // double distance = back_ground_estimation(in, bg_value, candidate_value, time_matrix);
    // int idx = y * in.width + x;
    // double distance = 0;
    if (distance < strictDistanceThreshold)
    {
        in.buffer[idx] = {0, 0, 0};
    }
    else
    {
        uint8_t intensity = static_cast<uint8_t>(mymin(255.0, distance * highlightDistanceMultiplier));
        in.buffer[idx] = {intensity, intensity, 0};
    }
    
}


void compute_cu(ImageView<rgb8> in) {
    hipError_t err;
    static bool initialized = false;
    static Image<rgb8> device_bg;
    static Image<rgb8> device_candidate;
    int* time_matrix;

    // Define thresholds for smoother filtering
    const float min_threshold = 10.0f;
    const float max_threshold = 20.0f;
    const int max_time_since_match = 50;

    // Allocate device memory for images and distances
    Image<rgb8> device_in(in.width, in.height, true);
    if (!initialized) {
        device_bg = Image<rgb8>(in.width, in.height, true);
        device_candidate = Image<rgb8>(in.width, in.height, true);
    }
    // float* distances;
    // CUDA_CHECK(hipMalloc(&distances, in.width * in.height * sizeof(float)));

    // Copy input image to device
    CUDA_CHECK(hipMemcpy2D(device_in.buffer, device_in.stride, in.buffer, in.stride,
                            in.width * sizeof(rgb8), in.height, hipMemcpyHostToDevice));

    if (!initialized) {
        // Initialize background and candidate images
        CUDA_CHECK(hipMemcpy(device_bg.buffer, device_in.buffer, in.width * in.height * sizeof(rgb8), hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipMemcpy(device_candidate.buffer, device_in.buffer, in.width * in.height * sizeof(rgb8), hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipMalloc(&time_matrix, in.width * in.height * sizeof(int)));
        initialized = true;
    }

    // Compute distances between background and input image
    dim3 block(16, 16);
    dim3 grid((in.width + block.x - 1) / block.x, (in.height + block.y - 1) / block.y);
    
    

    std::cout << "before apply" << std::endl;
    applyFlow<<<grid, block>>>(device_in, device_bg, device_candidate, time_matrix);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Erreur lors du lancement du filtre : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Synchroniser le dispositif
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Erreur lors de la synchronisation du dispositif : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copier le résultat vers l'hôte
    err = hipMemcpy2D(in.buffer, in.stride, device_in.buffer, device_in.stride, in.width * sizeof(rgb8), in.height, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Erreur lors de la copie de l'image traitée vers l'hôte : %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
        
    }
}
